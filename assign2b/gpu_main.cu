#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

// === microblock layout ===
int mb_rows;
int mb_cols;

//================================================================================
//======================== Main Driver ===========================================
//================================================================================

int main( int argc, char **argv ) {
  // Synchronize with the CUDA runtime
  hipDeviceSynchronize();

  // Display help menu if desired
  if( find_option( argc, argv, "-h" ) >= 0 ) {
    printf( "Options:\n" );
    printf( "-h to see this help\n" );
    printf( "-n <int> to set the number of particles\n" );
    printf( "-o <filename> to specify the output file name\n" );
    return 0;
  }

  // Output file parameter
  char *savename = read_string(argc, argv, "-o", NULL);
  FILE *fsave = savename ? fopen(savename, "w") : NULL;

  // Number of Particles Parameter
  int n = read_int(argc, argv, "-n", 1000);

  // Initialize CPU Particle List
  particle_t *particles;
  hipHostMalloc(&particles, n * sizeof(particle_t));
  double size = set_size(n);
  init_particles(n, particles);

  // Initialize GPU Particle List
  particle_t* gpu_particles;
  hipMalloc(&gpu_particles, n * sizeof(particle_t));
  hipDeviceSynchronize();

  // Determine number of blocks
  double mb_size = 2 * cutoff;
  mb_rows = (int)(size / mb_size);
  mb_cols = (int)(size / mb_size);

  // Initialize GPU Microblock List
  microblock* gpu_microblocks;
  hipMalloc(&gpu_microblocks, mb_rows * mb_cols * sizeof(microblock));

  // Initialize CPU Microblock List
  microblock* cpu_microblocks = (microblock*)malloc(mb_rows * mb_cols * sizeof(microblock));

  // Synchronize mallocs
  hipDeviceSynchronize();

  // Copy the particles to the GPU
  double copy_time = read_timer();
  hipMemcpy(gpu_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  copy_time = read_timer( ) - copy_time;

  // Thread Block Structure
  int blks_p = (n + NUM_THREADS - 1) / NUM_THREADS;
  int blks_mb = (mb_rows * mb_cols + NUM_THREADS - 1) / NUM_THREADS;

  // Distribute particles into microblocks
  distribute_gpu <<< blks_mb, NUM_THREADS >>> (gpu_microblocks, mb_rows, mb_cols, gpu_particles, n, size);
  hipDeviceSynchronize();

  // Start Simulation
  double simulation_time = read_timer();
  for(int step = 0; step < NSTEPS; step++) {    
    // Compute Forces
    compute_forces_gpu <<< blks_mb, NUM_THREADS >>> (gpu_microblocks, mb_rows, mb_cols, gpu_particles);
    hipDeviceSynchronize();
              
    // Move particles
    move_gpu <<< blks_p, NUM_THREADS >>> (gpu_particles, n, size);
    hipDeviceSynchronize();

    // Migrate particles
    migrate_particles_gpu <<< blks_mb, NUM_THREADS >>> (gpu_microblocks, mb_rows, mb_cols, gpu_particles, n, size);
    hipDeviceSynchronize();

    // If save desired      
    if( fsave && (step%SAVEFREQ) == 0 ) {
      // Copy the particles back to the CPU
      hipMemcpy(particles, gpu_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
      save(fsave, n, particles);
    }
  }

  // Calculate Time Passed
  hipDeviceSynchronize();
  simulation_time = read_timer() - simulation_time;

  // Output Statistics
  printf( "CPU-GPU copy time = %g seconds\n", copy_time);
  printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );

  // Free memory
  hipHostFree(particles);
  hipFree(gpu_particles);
  hipFree(gpu_microblocks);

  // Close File
  if(fsave) fclose(fsave);
    
  return 0;
}
